#include "hip/hip_runtime.h"

#include <OptiXToolkit/DemandLoading/Texture2D.h>
#include <cstdio>

__global__ void testKernel( demandLoading::DeviceContext* context, hipTextureObject_t tex, float x, float y, float2 ddx, float2 ddy, bool* isResident)
{
    float C = tex2DGrad<float>( *context, tex, x, y, ddx, ddy, isResident );
    printf( "C = %f\n", C );
}
